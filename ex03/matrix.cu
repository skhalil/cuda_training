
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
#include <stdio.h>
// Here you can set the device ID that was assigned to you
#define MYDEVICE 0


// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);
// Part 2 of 4: implement the kernel
__global__ void kernel( int *a, int dimx, int dimy ) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;  
  //printf("a[%d] = %d\n", i*dimx+j, i*dimx+j);
  a[i*dimx+j] = i*dimx + j;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main() {
    hipSetDevice(MYDEVICE);
// Part 1 and 4 of 4: set the dimensions of the matrix
    int dimx = 4;
    int dimy = 4;
    int num_bytes = dimx*dimy*sizeof(int);

    int *d_a=0, *h_a=0; // device and host pointers

    h_a = (int*)malloc(num_bytes);
    //allocate memory on the device
    hipMalloc((void**)&d_a, num_bytes);

    if( NULL==h_a || NULL==d_a ) {
        std::cerr << "couldn't allocate memory" << std::endl;
        return 1;
    }

    // Part 2 of 4: define grid and block size and launch the kernel
    dim3 grid, block;
    block.x = 2;
    block.y = 2;
    grid.x  = 2;
    grid.y  = 2;

    kernel<<<grid, block>>>( d_a, dimx, dimy );
    // block until the device has completed
    hipDeviceSynchronize();
    // check if kernel execution generated an error
    checkCUDAError("kernel execution");
    // device to host copy
    hipMemcpy(h_a, d_a, num_bytes, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");
    // verify the data returned to the host is correct
    for(int row=0; row<dimy; row++)
    {
        for(int col=0; col<dimx; col++) {
            //printf("h_a[%d] = %d\n", row * dimx + col, row * dimx + col);
            assert(h_a[row * dimx + col] == row * dimx + col);
        }  
    }
    // free host memory
    free( h_a );
    // free device memory
    hipFree( d_a );

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    std::cout << "Correct!" << std::endl;

    return 0;
}


void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        std::cerr << "Cuda error: " << msg << " " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }                         
}
