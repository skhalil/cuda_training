// includes, system

#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main() 
{
    hipSetDevice(MYDEVICE);
    // pointer and dimension for host memory
    int dimA = 8;
    float *h_a;

    // pointers for device memory
    float *d_a, *d_b;

    // allocate and initialize host memory
    // Bonus: try using cudaMallocHost in place of malloc
    // it has the same syntax as cudaMalloc, but it enables asynchronous copies
    h_a = (float *) malloc(dimA*sizeof(float));
    for (int i = 0; i<dimA; ++i)
    {
        h_a[i] = i;
    }

    // Part 1 of 5: allocate device memory
    size_t memSize = dimA*sizeof(float);
    hipMalloc(&d_a, memSize );
    hipMalloc(&d_b, memSize );

    // Part 2 of 5: host to device memory copy
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);

    // Part 3 of 5: device to device memory copy
    hipMemcpy(d_b, d_a, memSize, hipMemcpyDeviceToDevice);

    // clear host memory
    for (int i=0; i<dimA; ++i )
    {
        h_a[i] = 0.f;
    }

    // Part 4 of 5: device to host copy
    hipMemcpy(h_a, d_b, memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy calls");

    // verify the data on the host is correct
    for (int i=0; i<dimA; ++i)
    {
        assert(h_a[i] == (float) i);
    }

    // Part 5 of 5: free device memory pointers d_a and d_b
    hipFree(d_a);
    hipFree(d_b);

    // Check for any CUDA errors
    checkCUDAError("hipFree");

    // free host memory pointer h_a
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    std::cout << "Correct!" << std::endl;

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        std::cerr << "Cuda error: " << msg << " " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }                         
}
