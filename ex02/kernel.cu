
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <stdio.h>
// Here you can set the device ID that was assigned to you
#define MYDEVICE 0


// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

// Part 3 of 5: implement the kernel
__global__ void myFirstKernel(int *d_a)
{
  //printf("blockIdx.x = %d, blockDim.x = %d, threadIdx.x = %d\n", blockIdx.x, blockDim.x, threadIdx.x);
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  //d_a[i] = i;
  d_a[i] = blockIdx.x + threadIdx.x;  
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    hipSetDevice(MYDEVICE);
    // pointer for host memory
    int *h_a;

    // pointer for device memory
    int *d_a;

    // define grid and block size
    int numBlocks = 8;
    int numThreadsPerBlock = 8;

    // Part 1 of 5: allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc((void**)&d_a, memSize);

    // Part 2 of 5: configure and launch kernel
    dim3 dimGrid(8,1,1);
    dim3 dimBlock(8,1,1);
    myFirstKernel<<<dimGrid, dimBlock>>>(d_a);

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    checkCUDAError("kernel execution");

    // Part 4 of 5: device to host copy
    hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy");

    // Part 5 of 5: verify the data returned to the host is correct
    for (int i = 0; i <  8        ; ++i)
    {
        for (int j = 0; j <       8            ; ++j)
        {
            //printf("h_a[%d] = %d\n", i*numThreadsPerBlock + j, i*numThreadsPerBlock + j);
            //assert(h_a[i * numThreadsPerBlock + j] == i *numThreadsPerBlock + j);
            //printf("h_a[%d] = %d\n", i*numThreadsPerBlock + j, i + j);
            assert(h_a[i * numThreadsPerBlock + j] == i + j);
        }
    }

    // free device memory
    hipFree(d_a);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    std::cout << "Correct!" << std::endl;

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        std::cerr << "Cuda error: " << msg << " " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }                         
}
